#include "hip/hip_runtime.h"
/* =====================================
 *
 * Copyright (c) 2023, AUSTRAC Australian Government
 * All rights reserved.
 *
 * Licensed under BSD 3 clause license
 *
 */

/*
 * A direct translation to the GPU of the 32-bit libsodium primitives for Ed25519.
 *
 * Most of this code is a verbatim copy of
 *
 *   src/libsodium/crypto_core/ed25519/ref10/ed25519_ref10.c
 *
 * Minor changes have been made for #includes of certain precomputed tables.
 */

#include <stdint.h>

#include "fe_constants.h"

/*
 ge means group element.

 Here the group is the set of pairs (x,y) of field elements
 satisfying -x^2 + y^2 = 1 + d x^2y^2
 where d = -121665/121666.

 Representations:
 ge25519_p2 (projective): (X:Y:Z) satisfying x=X/Z, y=Y/Z
 ge25519_p3 (extended): (X:Y:Z:T) satisfying x=X/Z, y=Y/Z, XY=ZT
 ge25519_p1p1 (completed): ((X:Z),(Y:T)) satisfying x=X/Z, y=Y/T
 ge25519_precomp (Duif): (y+x,y-x,2dxy)
 */

typedef struct {
    fe25519 X;
    fe25519 Y;
    fe25519 Z;
} ge25519_p2;

typedef struct {
    fe25519 X;
    fe25519 Y;
    fe25519 Z;
    fe25519 T;
} ge25519_p3;

typedef struct {
    fe25519 X;
    fe25519 Y;
    fe25519 Z;
    fe25519 T;
} ge25519_p1p1;

typedef struct {
    fe25519 yplusx;
    fe25519 yminusx;
    fe25519 xy2d;
} ge25519_precomp;

typedef struct {
    fe25519 YplusX;
    fe25519 YminusX;
    fe25519 Z;
    fe25519 T2d;
} ge25519_cached;


/*
 r = p + q
 */

__host__ __device__ void
ge25519_add(ge25519_p1p1 *r, const ge25519_p3 *p, const ge25519_cached *q)
{
    fe25519 t0;

    fe25519_add(r->X, p->Y, p->X);
    fe25519_sub(r->Y, p->Y, p->X);
    fe25519_mul(r->Z, r->X, q->YplusX);
    fe25519_mul(r->Y, r->Y, q->YminusX);
    fe25519_mul(r->T, q->T2d, p->T);
    fe25519_mul(r->X, p->Z, q->Z);
    fe25519_add(t0, r->X, r->X);
    fe25519_sub(r->X, r->Z, r->Y);
    fe25519_add(r->Y, r->Z, r->Y);
    fe25519_add(r->Z, t0, r->T);
    fe25519_sub(r->T, t0, r->T);
}

static __host__ __device__ void
slide_vartime(signed char *r, const unsigned char *a)
{
    int i;
    int b;
    int k;
    int ribs;
    int cmp;

    for (i = 0; i < 256; ++i) {
        r[i] = 1 & (a[i >> 3] >> (i & 7));
    }
    for (i = 0; i < 256; ++i) {
        if (! r[i]) {
            continue;
        }
        for (b = 1; b <= 6 && i + b < 256; ++b) {
            if (! r[i + b]) {
                continue;
            }
            ribs = r[i + b] << b;
            cmp = r[i] + ribs;
            if (cmp <= 15) {
                r[i] = cmp;
                r[i + b] = 0;
            } else {
                cmp = r[i] - ribs;
                if (cmp < -15) {
                    break;
                }
                r[i] = cmp;
                for (k = i + b; k < 256; ++k) {
                    if (! r[k]) {
                        r[k] = 1;
                        break;
                    }
                    r[k] = 0;
                }
            }
        }
    }
}

__host__ __device__ int
ge25519_frombytes(ge25519_p3 *h, const unsigned char *s)
{
    fe25519 u;
    fe25519 v;
    fe25519 v3;
    fe25519 vxx;
    fe25519 m_root_check, p_root_check;
    fe25519 negx;
    fe25519 x_sqrtm1;
    int     has_m_root, has_p_root;

    fe25519_frombytes(h->Y, s);
    fe25519_1(h->Z);
    fe25519_sq(u, h->Y);
    fe25519_mul(v, u, d);
    fe25519_sub(u, u, h->Z); /* u = y^2-1 */
    fe25519_add(v, v, h->Z); /* v = dy^2+1 */

    fe25519_sq(v3, v);
    fe25519_mul(v3, v3, v); /* v3 = v^3 */
    fe25519_sq(h->X, v3);
    fe25519_mul(h->X, h->X, v);
    fe25519_mul(h->X, h->X, u); /* x = uv^7 */

    fe25519_pow22523(h->X, h->X); /* x = (uv^7)^((q-5)/8) */
    fe25519_mul(h->X, h->X, v3);
    fe25519_mul(h->X, h->X, u); /* x = uv^3(uv^7)^((q-5)/8) */

    fe25519_sq(vxx, h->X);
    fe25519_mul(vxx, vxx, v);
    fe25519_sub(m_root_check, vxx, u); /* vx^2-u */
    fe25519_add(p_root_check, vxx, u); /* vx^2+u */
    has_m_root = fe25519_iszero(m_root_check);
    has_p_root = fe25519_iszero(p_root_check);
    fe25519_mul(x_sqrtm1, h->X, sqrtm1); /* x*sqrt(-1) */
    fe25519_cmov(h->X, x_sqrtm1, 1 - has_m_root);

    fe25519_neg(negx, h->X);
    fe25519_cmov(h->X, negx, fe25519_isnegative(h->X) ^ (s[31] >> 7));
    fe25519_mul(h->T, h->X, h->Y);

    return (has_m_root | has_p_root) - 1;
}

__host__ __device__ int
ge25519_frombytes_negate_vartime(ge25519_p3 *h, const unsigned char *s)
{
    fe25519 u;
    fe25519 v;
    fe25519 v3;
    fe25519 vxx;
    fe25519 m_root_check, p_root_check;

    fe25519_frombytes(h->Y, s);
    fe25519_1(h->Z);
    fe25519_sq(u, h->Y);
    fe25519_mul(v, u, d);
    fe25519_sub(u, u, h->Z); /* u = y^2-1 */
    fe25519_add(v, v, h->Z); /* v = dy^2+1 */

    fe25519_sq(v3, v);
    fe25519_mul(v3, v3, v); /* v3 = v^3 */
    fe25519_sq(h->X, v3);
    fe25519_mul(h->X, h->X, v);
    fe25519_mul(h->X, h->X, u); /* x = uv^7 */

    fe25519_pow22523(h->X, h->X); /* x = (uv^7)^((q-5)/8) */
    fe25519_mul(h->X, h->X, v3);
    fe25519_mul(h->X, h->X, u); /* x = uv^3(uv^7)^((q-5)/8) */

    fe25519_sq(vxx, h->X);
    fe25519_mul(vxx, vxx, v);
    fe25519_sub(m_root_check, vxx, u); /* vx^2-u */
    if (fe25519_iszero(m_root_check) == 0) {
        fe25519_add(p_root_check, vxx, u); /* vx^2+u */
        if (fe25519_iszero(p_root_check) == 0) {
            return -1;
        }
        fe25519_mul(h->X, h->X, sqrtm1);
    }

    if (fe25519_isnegative(h->X) == (s[31] >> 7)) {
        fe25519_neg(h->X, h->X);
    }
    fe25519_mul(h->T, h->X, h->Y);

    return 0;
}

/*
 r = p + q
 */

static __host__ __device__ void
ge25519_madd(ge25519_p1p1 *r, const ge25519_p3 *p, const ge25519_precomp *q)
{
    fe25519 t0;

    fe25519_add(r->X, p->Y, p->X);
    fe25519_sub(r->Y, p->Y, p->X);
    fe25519_mul(r->Z, r->X, q->yplusx);
    fe25519_mul(r->Y, r->Y, q->yminusx);
    fe25519_mul(r->T, q->xy2d, p->T);
    fe25519_add(t0, p->Z, p->Z);
    fe25519_sub(r->X, r->Z, r->Y);
    fe25519_add(r->Y, r->Z, r->Y);
    fe25519_add(r->Z, t0, r->T);
    fe25519_sub(r->T, t0, r->T);
}

/*
 r = p - q
 */

static __host__ __device__ void
ge25519_msub(ge25519_p1p1 *r, const ge25519_p3 *p, const ge25519_precomp *q)
{
    fe25519 t0;

    fe25519_add(r->X, p->Y, p->X);
    fe25519_sub(r->Y, p->Y, p->X);
    fe25519_mul(r->Z, r->X, q->yminusx);
    fe25519_mul(r->Y, r->Y, q->yplusx);
    fe25519_mul(r->T, q->xy2d, p->T);
    fe25519_add(t0, p->Z, p->Z);
    fe25519_sub(r->X, r->Z, r->Y);
    fe25519_add(r->Y, r->Z, r->Y);
    fe25519_sub(r->Z, t0, r->T);
    fe25519_add(r->T, t0, r->T);
}

/*
 r = p
 */

__host__ __device__ void
ge25519_p1p1_to_p2(ge25519_p2 *r, const ge25519_p1p1 *p)
{
    fe25519_mul(r->X, p->X, p->T);
    fe25519_mul(r->Y, p->Y, p->Z);
    fe25519_mul(r->Z, p->Z, p->T);
}

/*
 r = p
 */

__host__ __device__ void
ge25519_p1p1_to_p3(ge25519_p3 *r, const ge25519_p1p1 *p)
{
    fe25519_mul(r->X, p->X, p->T);
    fe25519_mul(r->Y, p->Y, p->Z);
    fe25519_mul(r->Z, p->Z, p->T);
    fe25519_mul(r->T, p->X, p->Y);
}

static __host__ __device__ void
ge25519_p2_0(ge25519_p2 *h)
{
    fe25519_0(h->X);
    fe25519_1(h->Y);
    fe25519_1(h->Z);
}

/*
 r = 2 * p
 */

static __host__ __device__ void
ge25519_p2_dbl(ge25519_p1p1 *r, const ge25519_p2 *p)
{
    fe25519 t0;

    fe25519_sq(r->X, p->X);
    fe25519_sq(r->Z, p->Y);
    fe25519_sq2(r->T, p->Z);
    fe25519_add(r->Y, p->X, p->Y);
    fe25519_sq(t0, r->Y);
    fe25519_add(r->Y, r->Z, r->X);
    fe25519_sub(r->Z, r->Z, r->X);
    fe25519_sub(r->X, t0, r->Y);
    fe25519_sub(r->T, r->T, r->Z);
}

static __host__ __device__ void
ge25519_p3_0(ge25519_p3 *h)
{
    fe25519_0(h->X);
    fe25519_1(h->Y);
    fe25519_1(h->Z);
    fe25519_0(h->T);
}

static __host__ __device__ void
ge25519_cached_0(ge25519_cached *h)
{
    fe25519_1(h->YplusX);
    fe25519_1(h->YminusX);
    fe25519_1(h->Z);
    fe25519_0(h->T2d);
}

/*
 r = p
 */

__host__ __device__ void
ge25519_p3_to_cached(ge25519_cached *r, const ge25519_p3 *p)
{
    fe25519_add(r->YplusX, p->Y, p->X);
    fe25519_sub(r->YminusX, p->Y, p->X);
    fe25519_copy(r->Z, p->Z);
    fe25519_mul(r->T2d, p->T, d2);
}

static __host__ __device__ void
ge25519_p3_to_precomp(ge25519_precomp *pi, const ge25519_p3 *p)
{
    fe25519 recip;
    fe25519 x;
    fe25519 y;
    fe25519 xy;

    fe25519_invert(recip, p->Z);
    fe25519_mul(x, p->X, recip);
    fe25519_mul(y, p->Y, recip);
    fe25519_add(pi->yplusx, y, x);
    fe25519_sub(pi->yminusx, y, x);
    fe25519_mul(xy, x, y);
    fe25519_mul(pi->xy2d, xy, d2);
}

/*
 r = p
 */

static __host__ __device__ void
ge25519_p3_to_p2(ge25519_p2 *r, const ge25519_p3 *p)
{
    fe25519_copy(r->X, p->X);
    fe25519_copy(r->Y, p->Y);
    fe25519_copy(r->Z, p->Z);
}

__host__ __device__ void
ge25519_p3_tobytes(unsigned char *s, const ge25519_p3 *h)
{
    fe25519 recip;
    fe25519 x;
    fe25519 y;

    fe25519_invert(recip, h->Z);
    fe25519_mul(x, h->X, recip);
    fe25519_mul(y, h->Y, recip);
    fe25519_tobytes(s, y);
    s[31] ^= fe25519_isnegative(x) << 7;
}

/*
 r = 2 * p
 */

static __host__ __device__ void
ge25519_p3_dbl(ge25519_p1p1 *r, const ge25519_p3 *p)
{
    ge25519_p2 q;
    ge25519_p3_to_p2(&q, p);
    ge25519_p2_dbl(r, &q);
}

static __host__ __device__ void
ge25519_precomp_0(ge25519_precomp *h)
{
    fe25519_1(h->yplusx);
    fe25519_1(h->yminusx);
    fe25519_0(h->xy2d);
}

static __host__ __device__ unsigned char
equal(signed char b, signed char c)
{
    unsigned char ub = b;
    unsigned char uc = c;
    unsigned char x  = ub ^ uc; /* 0: yes; 1..255: no */
    uint32_t      y  = (uint32_t) x; /* 0: yes; 1..255: no */

    y -= 1;   /* 4294967295: yes; 0..254: no */
    y >>= 31; /* 1: yes; 0: no */

    return y;
}

static __host__ __device__ unsigned char
negative(signed char b)
{
    /* 18446744073709551361..18446744073709551615: yes; 0..255: no */
    uint64_t x = b;

    x >>= 63; /* 1: yes; 0: no */

    return x;
}

static __host__ __device__ void
ge25519_cmov(ge25519_precomp *t, const ge25519_precomp *u, unsigned char b)
{
    fe25519_cmov(t->yplusx, u->yplusx, b);
    fe25519_cmov(t->yminusx, u->yminusx, b);
    fe25519_cmov(t->xy2d, u->xy2d, b);
}

static __host__ __device__ void
ge25519_cmov_cached(ge25519_cached *t, const ge25519_cached *u, unsigned char b)
{
    fe25519_cmov(t->YplusX, u->YplusX, b);
    fe25519_cmov(t->YminusX, u->YminusX, b);
    fe25519_cmov(t->Z, u->Z, b);
    fe25519_cmov(t->T2d, u->T2d, b);
}

static __host__ __device__ void
ge25519_cmov8(ge25519_precomp *t, const ge25519_precomp precomp[8], const signed char b)
{
    ge25519_precomp     minust;
    const unsigned char bnegative = negative(b);
    const unsigned char babs      = b - (((-bnegative) & b) * ((signed char) 1 << 1));

    ge25519_precomp_0(t);
    ge25519_cmov(t, &precomp[0], equal(babs, 1));
    ge25519_cmov(t, &precomp[1], equal(babs, 2));
    ge25519_cmov(t, &precomp[2], equal(babs, 3));
    ge25519_cmov(t, &precomp[3], equal(babs, 4));
    ge25519_cmov(t, &precomp[4], equal(babs, 5));
    ge25519_cmov(t, &precomp[5], equal(babs, 6));
    ge25519_cmov(t, &precomp[6], equal(babs, 7));
    ge25519_cmov(t, &precomp[7], equal(babs, 8));
    fe25519_copy(minust.yplusx, t->yminusx);
    fe25519_copy(minust.yminusx, t->yplusx);
    fe25519_neg(minust.xy2d, t->xy2d);
    ge25519_cmov(t, &minust, bnegative);
}

static const ge25519_precomp __device__ _ge25519_cmov8_base_precomp[32][8] = { /* base[i][j] = (j+1)*256^i*B */
#include "ge25519_cmov8_base.h"
};

static __host__ __device__ void
ge25519_cmov8_base(ge25519_precomp *t, const int pos, const signed char b)
{
    ge25519_cmov8(t, _ge25519_cmov8_base_precomp[pos], b);
}

static __host__ __device__ void
ge25519_cmov8_cached(ge25519_cached *t, const ge25519_cached cached[8], const signed char b)
{
    ge25519_cached      minust;
    const unsigned char bnegative = negative(b);
    const unsigned char babs      = b - (((-bnegative) & b) * ((signed char) 1 << 1));

    ge25519_cached_0(t);
    ge25519_cmov_cached(t, &cached[0], equal(babs, 1));
    ge25519_cmov_cached(t, &cached[1], equal(babs, 2));
    ge25519_cmov_cached(t, &cached[2], equal(babs, 3));
    ge25519_cmov_cached(t, &cached[3], equal(babs, 4));
    ge25519_cmov_cached(t, &cached[4], equal(babs, 5));
    ge25519_cmov_cached(t, &cached[5], equal(babs, 6));
    ge25519_cmov_cached(t, &cached[6], equal(babs, 7));
    ge25519_cmov_cached(t, &cached[7], equal(babs, 8));
    fe25519_copy(minust.YplusX, t->YminusX);
    fe25519_copy(minust.YminusX, t->YplusX);
    fe25519_copy(minust.Z, t->Z);
    fe25519_neg(minust.T2d, t->T2d);
    ge25519_cmov_cached(t, &minust, bnegative);
}

/*
 r = p - q
 */

__host__ __device__ void
ge25519_sub(ge25519_p1p1 *r, const ge25519_p3 *p, const ge25519_cached *q)
{
    fe25519 t0;

    fe25519_add(r->X, p->Y, p->X);
    fe25519_sub(r->Y, p->Y, p->X);
    fe25519_mul(r->Z, r->X, q->YminusX);
    fe25519_mul(r->Y, r->Y, q->YplusX);
    fe25519_mul(r->T, q->T2d, p->T);
    fe25519_mul(r->X, p->Z, q->Z);
    fe25519_add(t0, r->X, r->X);
    fe25519_sub(r->X, r->Z, r->Y);
    fe25519_add(r->Y, r->Z, r->Y);
    fe25519_sub(r->Z, t0, r->T);
    fe25519_add(r->T, t0, r->T);
}

__host__ __device__ void
ge25519_tobytes(unsigned char *s, const ge25519_p2 *h)
{
    fe25519 recip;
    fe25519 x;
    fe25519 y;

    fe25519_invert(recip, h->Z);
    fe25519_mul(x, h->X, recip);
    fe25519_mul(y, h->Y, recip);
    fe25519_tobytes(s, y);
    s[31] ^= fe25519_isnegative(x) << 7;
}

/*
 r = a * A + b * B
 where a = a[0]+256*a[1]+...+256^31 a[31].
 and b = b[0]+256*b[1]+...+256^31 b[31].
 B is the Ed25519 base point (x,4/5) with x positive.

 Only used for signatures verification.
 */

__host__ __device__ void
ge25519_double_scalarmult_vartime(ge25519_p2 *r, const unsigned char *a,
                                  const ge25519_p3 *A, const unsigned char *b)
{
    static const ge25519_precomp Bi[8] = {
#include "ge25519_double_scalarmult_vartime_base2.h"
    };
    signed char    aslide[256];
    signed char    bslide[256];
    ge25519_cached Ai[8]; /* A,3A,5A,7A,9A,11A,13A,15A */
    ge25519_p1p1   t;
    ge25519_p3     u;
    ge25519_p3     A2;
    int            i;

    slide_vartime(aslide, a);
    slide_vartime(bslide, b);

    ge25519_p3_to_cached(&Ai[0], A);

    ge25519_p3_dbl(&t, A);
    ge25519_p1p1_to_p3(&A2, &t);

    ge25519_add(&t, &A2, &Ai[0]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[1], &u);

    ge25519_add(&t, &A2, &Ai[1]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[2], &u);

    ge25519_add(&t, &A2, &Ai[2]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[3], &u);

    ge25519_add(&t, &A2, &Ai[3]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[4], &u);

    ge25519_add(&t, &A2, &Ai[4]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[5], &u);

    ge25519_add(&t, &A2, &Ai[5]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[6], &u);

    ge25519_add(&t, &A2, &Ai[6]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[7], &u);

    ge25519_p2_0(r);

    for (i = 255; i >= 0; --i) {
        if (aslide[i] || bslide[i]) {
            break;
        }
    }

    for (; i >= 0; --i) {
        ge25519_p2_dbl(&t, r);

        if (aslide[i] > 0) {
            ge25519_p1p1_to_p3(&u, &t);
            ge25519_add(&t, &u, &Ai[aslide[i] / 2]);
        } else if (aslide[i] < 0) {
            ge25519_p1p1_to_p3(&u, &t);
            ge25519_sub(&t, &u, &Ai[(-aslide[i]) / 2]);
        }

        if (bslide[i] > 0) {
            ge25519_p1p1_to_p3(&u, &t);
            ge25519_madd(&t, &u, &Bi[bslide[i] / 2]);
        } else if (bslide[i] < 0) {
            ge25519_p1p1_to_p3(&u, &t);
            ge25519_msub(&t, &u, &Bi[(-bslide[i]) / 2]);
        }

        ge25519_p1p1_to_p2(r, &t);
    }
}

/*
 h = a * p
 where a = a[0]+256*a[1]+...+256^31 a[31]

 Preconditions:
 a[31] <= 127

 p is public
 */

__host__ __device__ void
ge25519_scalarmult(ge25519_p3 *h, const unsigned char *a, const ge25519_p3 *p)
{
    signed char     e[64];
    signed char     carry;
    ge25519_p1p1    r;
    ge25519_p2      s;
    ge25519_p1p1    t2, t3, t4, t5, t6, t7, t8;
    ge25519_p3      p2, p3, p4, p5, p6, p7, p8;
    ge25519_cached  pi[8];
    ge25519_cached  t;
    int             i;

    ge25519_p3_to_cached(&pi[1 - 1], p);   /* p */

    ge25519_p3_dbl(&t2, p);
    ge25519_p1p1_to_p3(&p2, &t2);
    ge25519_p3_to_cached(&pi[2 - 1], &p2); /* 2p = 2*p */

    ge25519_add(&t3, p, &pi[2 - 1]);
    ge25519_p1p1_to_p3(&p3, &t3);
    ge25519_p3_to_cached(&pi[3 - 1], &p3); /* 3p = 2p+p */

    ge25519_p3_dbl(&t4, &p2);
    ge25519_p1p1_to_p3(&p4, &t4);
    ge25519_p3_to_cached(&pi[4 - 1], &p4); /* 4p = 2*2p */

    ge25519_add(&t5, p, &pi[4 - 1]);
    ge25519_p1p1_to_p3(&p5, &t5);
    ge25519_p3_to_cached(&pi[5 - 1], &p5); /* 5p = 4p+p */

    ge25519_p3_dbl(&t6, &p3);
    ge25519_p1p1_to_p3(&p6, &t6);
    ge25519_p3_to_cached(&pi[6 - 1], &p6); /* 6p = 2*3p */

    ge25519_add(&t7, p, &pi[6 - 1]);
    ge25519_p1p1_to_p3(&p7, &t7);
    ge25519_p3_to_cached(&pi[7 - 1], &p7); /* 7p = 6p+p */

    ge25519_p3_dbl(&t8, &p4);
    ge25519_p1p1_to_p3(&p8, &t8);
    ge25519_p3_to_cached(&pi[8 - 1], &p8); /* 8p = 2*4p */

    for (i = 0; i < 32; ++i) {
        e[2 * i + 0] = (a[i] >> 0) & 15;
        e[2 * i + 1] = (a[i] >> 4) & 15;
    }
    /* each e[i] is between 0 and 15 */
    /* e[63] is between 0 and 7 */

    carry = 0;
    for (i = 0; i < 63; ++i) {
        e[i] += carry;
        carry = e[i] + 8;
        carry >>= 4;
        e[i] -= carry * ((signed char) 1 << 4);
    }
    e[63] += carry;
    /* each e[i] is between -8 and 8 */

    ge25519_p3_0(h);

    for (i = 63; i != 0; i--) {
        ge25519_cmov8_cached(&t, pi, e[i]);
        ge25519_add(&r, h, &t);

        ge25519_p1p1_to_p2(&s, &r);
        ge25519_p2_dbl(&r, &s);
        ge25519_p1p1_to_p2(&s, &r);
        ge25519_p2_dbl(&r, &s);
        ge25519_p1p1_to_p2(&s, &r);
        ge25519_p2_dbl(&r, &s);
        ge25519_p1p1_to_p2(&s, &r);
        ge25519_p2_dbl(&r, &s);

        ge25519_p1p1_to_p3(h, &r);  /* *16 */
    }
    ge25519_cmov8_cached(&t, pi, e[i]);
    ge25519_add(&r, h, &t);

    ge25519_p1p1_to_p3(h, &r);
}

/*
 h = a * B (with precomputation)
 where a = a[0]+256*a[1]+...+256^31 a[31]
 B is the Ed25519 base point (x,4/5) with x positive
 (as bytes: 0x5866666666666666666666666666666666666666666666666666666666666666)

 Preconditions:
 a[31] <= 127
 */

__host__ __device__ void
ge25519_scalarmult_base(ge25519_p3 *h, const unsigned char *a)
{
    signed char     e[64];
    signed char     carry;
    ge25519_p1p1    r;
    ge25519_p2      s;
    ge25519_precomp t;
    int             i;

    for (i = 0; i < 32; ++i) {
        e[2 * i + 0] = (a[i] >> 0) & 15;
        e[2 * i + 1] = (a[i] >> 4) & 15;
    }
    /* each e[i] is between 0 and 15 */
    /* e[63] is between 0 and 7 */

    carry = 0;
    for (i = 0; i < 63; ++i) {
        e[i] += carry;
        carry = e[i] + 8;
        carry >>= 4;
        e[i] -= carry * ((signed char) 1 << 4);
    }
    e[63] += carry;
    /* each e[i] is between -8 and 8 */

    ge25519_p3_0(h);

    for (i = 1; i < 64; i += 2) {
        ge25519_cmov8_base(&t, i / 2, e[i]);
        ge25519_madd(&r, h, &t);
        ge25519_p1p1_to_p3(h, &r);
    }

    ge25519_p3_dbl(&r, h);
    ge25519_p1p1_to_p2(&s, &r);
    ge25519_p2_dbl(&r, &s);
    ge25519_p1p1_to_p2(&s, &r);
    ge25519_p2_dbl(&r, &s);
    ge25519_p1p1_to_p2(&s, &r);
    ge25519_p2_dbl(&r, &s);
    ge25519_p1p1_to_p3(h, &r);

    for (i = 0; i < 64; i += 2) {
        ge25519_cmov8_base(&t, i / 2, e[i]);
        ge25519_madd(&r, h, &t);
        ge25519_p1p1_to_p3(h, &r);
    }
}

/* multiply by the order of the main subgroup l = 2^252+27742317777372353535851937790883648493 */
static __host__ __device__ void
ge25519_mul_l(ge25519_p3 *r, const ge25519_p3 *A)
{
    static const signed char aslide[253] = {
        13, 0, 0, 0, 0, -1, 0, 0, 0, 0, -11, 0, 0, 0, 0, 0, 0, -5, 0, 0, 0, 0, 0, 0, -3, 0, 0, 0, 0, -13, 0, 0, 0, 0, 7, 0, 0, 0, 0, 0, 3, 0, 0, 0, 0, -13, 0, 0, 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 0, 11, 0, 0, 0, 0, 0, 11, 0, 0, 0, 0, -13, 0, 0, 0, 0, 0, 0, -3, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 3, 0, 0, 0, 0, -11, 0, 0, 0, 0, 0, 0, 0, 15, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, -1, 0, 0, 0, 0, 7, 0, 0, 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1
    };
    ge25519_cached Ai[8];
    ge25519_p1p1   t;
    ge25519_p3     u;
    ge25519_p3     A2;
    int            i;

    ge25519_p3_to_cached(&Ai[0], A);
    ge25519_p3_dbl(&t, A);
    ge25519_p1p1_to_p3(&A2, &t);
    ge25519_add(&t, &A2, &Ai[0]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[1], &u);
    ge25519_add(&t, &A2, &Ai[1]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[2], &u);
    ge25519_add(&t, &A2, &Ai[2]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[3], &u);
    ge25519_add(&t, &A2, &Ai[3]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[4], &u);
    ge25519_add(&t, &A2, &Ai[4]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[5], &u);
    ge25519_add(&t, &A2, &Ai[5]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[6], &u);
    ge25519_add(&t, &A2, &Ai[6]);
    ge25519_p1p1_to_p3(&u, &t);
    ge25519_p3_to_cached(&Ai[7], &u);

    ge25519_p3_0(r);

    for (i = 252; i >= 0; --i) {
        ge25519_p3_dbl(&t, r);

        if (aslide[i] > 0) {
            ge25519_p1p1_to_p3(&u, &t);
            ge25519_add(&t, &u, &Ai[aslide[i] / 2]);
        } else if (aslide[i] < 0) {
            ge25519_p1p1_to_p3(&u, &t);
            ge25519_sub(&t, &u, &Ai[(-aslide[i]) / 2]);
        }

        ge25519_p1p1_to_p3(r, &t);
    }
}

__host__ __device__ int
ge25519_is_on_curve(const ge25519_p3 *p)
{
    fe25519 x2;
    fe25519 y2;
    fe25519 z2;
    fe25519 z4;
    fe25519 t0;
    fe25519 t1;

    fe25519_sq(x2, p->X);
    fe25519_sq(y2, p->Y);
    fe25519_sq(z2, p->Z);
    fe25519_sub(t0, y2, x2);
    fe25519_mul(t0, t0, z2);

    fe25519_mul(t1, x2, y2);
    fe25519_mul(t1, t1, d);
    fe25519_sq(z4, z2);
    fe25519_add(t1, t1, z4);
    fe25519_sub(t0, t0, t1);

    return fe25519_iszero(t0);
}

__host__ __device__ int
ge25519_is_on_main_subgroup(const ge25519_p3 *p)
{
    ge25519_p3 pl;

    ge25519_mul_l(&pl, p);

    return fe25519_iszero(pl.X);
}

__host__ __device__ int
ge25519_is_canonical(const unsigned char *s)
{
    unsigned char c;
    unsigned char d;
    unsigned int  i;

    c = (s[31] & 0x7f) ^ 0x7f;
    for (i = 30; i > 0; i--) {
        c |= s[i] ^ 0xff;
    }
    c = (((unsigned int) c) - 1U) >> 8;
    d = (0xed - 1U - (unsigned int) s[0]) >> 8;

    return 1 - (c & d & 1);
}
